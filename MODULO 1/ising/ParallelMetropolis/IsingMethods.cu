#include "hip/hip_runtime.h"
#include "IsingMethods.h"

/*********************************************
/********* COPY TO DISK FUNCTION *************
/********************************************/

__host__ void copyToDisk(int i, int batchSize, int memSize, signed short* dMagMarkov, signed short* dEnergyMarkov)
{
    std::string fileName = "batch" + std::to_string(i) + ".root";
    TFile file = TFile(fileName.c_str(), "recreate");
    TTree tree = TTree("data", "markov step data");

    signed short currentMag;
    signed short currentEnergy;
    unsigned short simIdx;

    tree.Branch("simIdx", &simIdx, "simIdx/s");
    tree.Branch("magMarkov", &currentMag, "magMarkov/S");
    tree.Branch("energyMarkov", &currentEnergy, "energyMarkov/S");

    for(int n=0; n<memSize;n++)
    {
        simIdx = n/batchSize;
        currentMag = dMagMarkov[n];
        currentEnergy = dEnergyMarkov[n];
        tree.Fill();
    }
    tree.Write();
    file.Close();
}


/*********************************************
/********* SIMULATION KERNEL *****************
/********************************************/

__global__ void simulation(int batchNum, int batchSize, double *dBeta, signed short* dMagMarkov, signed short* dEnergyMarkov, signed short* dMagCumulant, signed short* dEnergyCumulant, bool* dLattices, hiprandState* dStates)
{
    //get thread index
    unsigned short idx = threadIdx.x + blockIdx.x*blockDim.x;

    //get simulation parameters and random generator from the thread and block index
    double beta = 0.44 + 0.1*pow(2*(double)threadIdx.x/blockDim.x - 1,3);
    unsigned char L = 20 + blockIdx.x*10;

    //initialize inter-batch arrays if this is the first run
    if(batchNum == 0)
    {
            dMagCumulant[idx] = L*L;
            dEnergyCumulant[idx] = -2*L*L;
    }

    //get values from inter-batch arrays
    hiprandState localState = dStates[idx];
    signed short dMag = dMagCumulant[idx];
    signed short dEnergy = dEnergyCumulant[idx];

    //make a copy from global memory of the local lattice
    int latticeIdx=0;
    for(unsigned char i=0; i<blockIdx.x; i++)
        latticeIdx += pow((20+i*10),2)*blockDim.x;

    latticeIdx += L*L*threadIdx.x;
    bool* localLattice = &dLattices[latticeIdx];

    //declare useful variables
    int i0, j0;
    float acceptance;
    signed char force;

    //run metropolis algorithm
    for(unsigned long long i=0; i<100*batchSize; i++)
    {
        //take a random site
        randomLatticeSite(&localState, &i0, &j0);
        //evaluate "force"
        force = get(localLattice,i0-1,j0,L)+ get(localLattice,i0+1,j0,L) + get(localLattice,i0,j0-1,L) + get(localLattice,i0,j0+1,L);

        //calculate acceptance
        acceptance = min(float(1.), expf(-2*get(localLattice,i0,j0,L)*force*beta));

        //accept-reject step
        if(acceptance > hiprand_uniform(&localState))
        {
            dEnergy += get(localLattice,i0,j0,L)*force;
            dMag -= 2*get(localLattice,i0,j0,L);
            localLattice[flatten(i0,j0,L)] = !localLattice[flatten(i0,j0,L)];
        }

        //reduce data
        if(i%100==0)
        {
            dMagMarkov[idx*batchSize + i/100] = dMag;
            dEnergyMarkov[idx*batchSize + i/100] = dEnergy;
        }
    }

    //copy back current state of the random generator and inter-batch arrays
    dStates[idx] = localState;
    dMagCumulant[idx] = dMag;
    dEnergyCumulant[idx] = dEnergy;

    if(blockIdx.x==0)
        dBeta[threadIdx.x] = beta;
}



/*********************************************
/********** LATTICE FUNCTIONS ****************
/********************************************/

__host__ __device__ constexpr int flatten(int i, int j, int L)
{
    //periodic boundary conditions
    int m=i;
    int n=j;

    if(i<0)
        m = L-1;
    else if(i>L-1)
        m = 0;

    if(j<0)
        n = L-1;
    else if(j>L-1)
        n = 0;

    return m + n*L;
}


__device__ constexpr signed char convert(bool value)
{
    signed char res=-1;

    if(value)
        res=1;

    return res;
}


__device__ signed char get(bool* lattice, int i, int j, int L)
{
    bool value = lattice[flatten(i,j,L)];
    return convert(value);
}



/*********************************************
/******* RANDOM GENERATOR FUNCTIONS **********
/********************************************/

__global__ void initRandom(hiprandState *states, int seed)
{
    unsigned short simIdx = threadIdx.x + blockIdx.x*blockDim.x;

    //initialize random generator on each thread
    hiprand_init(seed, simIdx, 0, &states[simIdx]);
}


__device__ void randomLatticeSite(hiprandState *states, int *i0, int *j0){

    unsigned short simIdx = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned char L = 20 + blockIdx.x*10;
    hiprandState localState = states[simIdx];

    *i0 = (int)truncf(L*hiprand_uniform(&localState));
    *j0 = (int)truncf(L*hiprand_uniform(&localState));

    states[simIdx] = localState;
}
