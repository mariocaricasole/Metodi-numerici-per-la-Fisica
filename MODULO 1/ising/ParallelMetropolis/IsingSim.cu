#include "IsingMethods.h"
#include <thread>

int main()
{
    /************************ PARAMETERS ************************/
    //simulation parameters
    const unsigned char L = 4;
    const unsigned char nBeta = 64;
    const int dataPoints = 4e5;
    const int batchSize = 1e5;
    const unsigned long long memSize = L*nBeta*batchSize;

    /******************* MEMORY ALLOCATIONS *********************/
    //allocate random generators
    hiprandState *dStates;
    hipMallocManaged(&dStates, L*nBeta*sizeof(hiprandState));

    //allocate unified memory
    double *dBeta;
    signed short **dMagMarkov, **dEnergyMarkov;
    signed short *dMagCumulant, *dEnergyCumulant;
    bool* dLattices;

    //allocate twin memory registers for simultaneous execution and copy of batches, make a host register for later use (planned interleaved GPU bootstrapping and CPU copy to disk)
    hipHostMalloc(&dMagMarkov, 3*sizeof(signed short*));
    hipHostMalloc(&dEnergyMarkov, 3*sizeof(signed short*));

    for(int i=0;i<2;i++)
    {
        hipMallocManaged(&dMagMarkov[i], memSize*sizeof(signed short));
        hipMallocManaged(&dEnergyMarkov[i], memSize*sizeof(signed short));
    }
    hipHostMalloc(&dMagMarkov[2], memSize*sizeof(signed short));
    hipHostMalloc(&dEnergyMarkov[2], memSize*sizeof(signed short));

    //allocating array of cumulants for magnetization and energy for inter-batch use
    hipHostMalloc(&dMagCumulant, nBeta*L*sizeof(signed short));
    hipHostMalloc(&dEnergyCumulant, nBeta*L*sizeof(signed short));

    //allocating array of betas and lattices memory for inter-batch use
    hipMallocManaged(&dBeta, nBeta*sizeof(double));
    hipMallocManaged(&dLattices, 5400*nBeta*sizeof(bool));

    /*********************** INITIALIZATIONS ********************/
    //initialize all lattices to cold state
    for(unsigned int i=0; i<5400*nBeta; i++)
        dLattices[i]=true;

    //run random generator initialization kernel
    initRandom<<<L,nBeta>>>(dStates,1221);

    //create event to signal end of kernel execution
    hipEvent_t kernelDone;
    hipEventCreate(&kernelDone);

    //create two streams, one for copy the other for execution of kernel
    hipStream_t copyStream, executeStream;
    hipStreamCreate(&copyStream);
    hipStreamCreate(&executeStream);
//
    /***************************** SIMULATION RUN *************************************/
    //run the kernel using one memory, while copying data to disk on the other memory
    for(int i=0; i<dataPoints/batchSize;i++){
        if(i!=0)
        {
            //wait for kernel execution to be completed
            hipStreamWaitEvent(copyStream, kernelDone);
            //start asynchronous memory copy of previous simulated data while new kernel starts
            hipMemcpyAsync(dMagMarkov[2], dMagMarkov[(i-1)%2], memSize*sizeof(signed short), hipMemcpyDeviceToHost, copyStream);
            hipMemcpyAsync(dEnergyMarkov[2], dEnergyMarkov[(i-1)%2], memSize*sizeof(signed short), hipMemcpyDeviceToHost, copyStream);
        }
        simulation<<<L,nBeta, 0, executeStream>>>(i,batchSize, dBeta, dMagMarkov[i%2], dEnergyMarkov[i%2], dMagCumulant, dEnergyCumulant, dLattices, dStates);
        //event of kernel execution finished recorded
        hipEventRecord(kernelDone, executeStream);
        if(i!=0)
        {
            //synchronize CPU copy to the end of the asynchronous copy in order to start copy to disk
            hipStreamSynchronize(copyStream);
            copyToDisk(i-1, batchSize, memSize, dMagMarkov[2], dEnergyMarkov[2]);
        }
    }
    //copy the last batch produced
    hipStreamWaitEvent(copyStream, kernelDone);
    hipMemcpyAsync(dMagMarkov[2], dMagMarkov[(dataPoints/batchSize -1)%2], memSize*sizeof(signed short), hipMemcpyDeviceToHost, copyStream);
    hipMemcpyAsync(dEnergyMarkov[2], dEnergyMarkov[(dataPoints/batchSize -1)%2], memSize*sizeof(signed short), hipMemcpyDeviceToHost, copyStream);
    hipStreamSynchronize(copyStream);
    copyToDisk(dataPoints/batchSize-1, batchSize, memSize, dMagMarkov[2], dEnergyMarkov[2]);
    //synchronize devices
    hipDeviceSynchronize();

    /********************************* CLEAN UP ****************************************/
    //destroy streams
    hipStreamDestroy(copyStream);
    hipStreamDestroy(executeStream);

    //free device memory
    hipFree(dStates);
    hipFree(dBeta);
    for(int i=0;i<3;i++)
    {
        hipFree(dMagMarkov[i]);
        hipFree(dEnergyMarkov[i]);
    }
    hipFree(dMagMarkov);
    hipFree(dEnergyMarkov);
    hipFree(dMagCumulant);
    hipFree(dEnergyCumulant);
    hipFree(dLattices);


    return 0;
}
