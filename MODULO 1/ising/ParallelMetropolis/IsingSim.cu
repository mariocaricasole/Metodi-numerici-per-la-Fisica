#include "IsingMethods.h"
#include <thread>

int main()
{
    /************************ PARAMETERS ************************/
    //simulation parameters
    const unsigned char L = 4;
    const unsigned char nBeta = 64;
    const int dataPoints = 2e5;
    const int batchSize = 1e5;
    const unsigned long long memSize = L*nBeta*batchSize;

    /******************* MEMORY ALLOCATIONS *********************/
    //allocate random generators
    hiprandState *dStates;
    hipMallocManaged(&dStates, L*nBeta*sizeof(hiprandState));

    //allocate unified memory
    double *dBeta;
    signed char **dMagMarkov;
    signed short **dEnergyMarkov;
    bool* dLattices;

    hipHostMalloc(&dMagMarkov, 3*sizeof(signed char*));
    hipHostMalloc(&dEnergyMarkov, 3*sizeof(signed short*));

    for(int i=0;i<2;i++)
    {
        hipMallocManaged(&dMagMarkov[i], memSize*sizeof(signed char));
        hipMallocManaged(&dEnergyMarkov[i], memSize*sizeof(signed short));
    }
    hipHostMalloc(&dMagMarkov[2], memSize*sizeof(signed char));
    hipHostMalloc(&dEnergyMarkov[2], memSize*sizeof(signed short));
    hipMallocManaged(&dBeta, nBeta*sizeof(double));
    hipMallocManaged(&dLattices, 5400*nBeta*sizeof(bool));

    /*********************** INITIALIZATIONS ********************/
    //initialize all lattices to cold state
    for(unsigned int i=0; i<5400*nBeta; i++)
        dLattices[i]=true;

    //run random generator initialization kernel
    initRandom<<<L,nBeta>>>(dStates,1221);

    //create event to signal end of kernel execution
    hipEvent_t kernelDone;
    hipEventCreate(&kernelDone);

    //create two streams, one for copy the other for execution of kernel
    hipStream_t copyStream, executeStream;
    hipStreamCreate(&copyStream);
    hipStreamCreate(&executeStream);
//
    /***************************** SIMULATION RUN *************************************/
    //run the kernel using one memory, while copying data to disk on the other memory
    for(int i=0; i<dataPoints/batchSize;i++){
        if(i!=0)
        {
            hipStreamWaitEvent(copyStream, kernelDone);
            hipMemcpyAsync(dMagMarkov[2], dMagMarkov[(i-1)%2], memSize*sizeof(signed char), hipMemcpyDeviceToHost, copyStream);
            hipMemcpyAsync(dEnergyMarkov[2], dEnergyMarkov[(i-1)%2], memSize*sizeof(signed short), hipMemcpyDeviceToHost, copyStream);
        }
        simulation<<<L,nBeta, 0, executeStream>>>(batchSize, dBeta, dMagMarkov[i%2], dEnergyMarkov[i%2], dLattices, dStates);
        hipEventRecord(kernelDone, executeStream);
        if(i!=0)
        {
            hipStreamSynchronize(copyStream);
            copyToDisk(i-1, batchSize, memSize, dMagMarkov[2], dEnergyMarkov[2]);
        }
    }
    //last copy
    hipStreamWaitEvent(copyStream, kernelDone);
    hipMemcpyAsync(dMagMarkov[2], dMagMarkov[(dataPoints/batchSize -1)%2], memSize*sizeof(signed char), hipMemcpyDeviceToHost, copyStream);
    hipMemcpyAsync(dEnergyMarkov[2], dEnergyMarkov[(dataPoints/batchSize -1)%2], memSize*sizeof(signed short), hipMemcpyDeviceToHost, copyStream);
    hipStreamSynchronize(copyStream);
    copyToDisk(dataPoints/batchSize-1, batchSize, memSize, dMagMarkov[2], dEnergyMarkov[2]);
    hipDeviceSynchronize();

    /********************************* CLEAN UP ****************************************/
    //destroy streams
    hipStreamDestroy(copyStream);
    hipStreamDestroy(executeStream);

    //free device memory
    hipFree(dStates);
    hipFree(dBeta);
    for(int i=0;i<3;i++)
    {
        hipFree(dMagMarkov[i]);
        hipFree(dEnergyMarkov[i]);
    }
    hipFree(dMagMarkov);
    hipFree(dEnergyMarkov);
    hipFree(dLattices);


    return 0;
}
