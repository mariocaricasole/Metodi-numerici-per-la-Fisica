#include "IsingMethods.h"
#include <thread>

int main()
{
    /************************ PARAMETERS ************************/
    //simulation parameters
    const unsigned char L = 4;
    const unsigned char nBeta = 64;
    const unsigned char nResamplings = 64;
    const int dataPoints = 1e5;
    const int batchSize = 1e5;
    const int blockingSize = 1e4;
    const unsigned long long memSize = L*nBeta*batchSize;

    /******************* MEMORY ALLOCATIONS *********************/
    //allocate random generators
    hiprandState *dStates, *dStatesBS;
    hipMallocManaged(&dStates, L*nBeta*sizeof(hiprandState));
    hipMallocManaged(&dStatesBS, L*nBeta*sizeof(hiprandState));

    //allocate unified memory
    double *dBeta;
    signed short **dMagMarkov, **dEnergyMarkov;
    signed short *dMagCurrent, *dEnergyCurrent;
    long long *dMagCumulant, *dEnergyCumulant;
    bool* dLattices;

    //allocate twin memory registers for simultaneous execution and copy of batches, make a host register for later use (planned interleaved GPU bootstrapping and CPU copy to disk)
    hipHostMalloc(&dMagMarkov, 2*sizeof(signed short*));
    hipHostMalloc(&dEnergyMarkov, 2*sizeof(signed short*));

    for(int i=0;i<2;i++)
    {
        hipMallocManaged(&dMagMarkov[i], memSize*sizeof(signed short));
        hipMallocManaged(&dEnergyMarkov[i], memSize*sizeof(signed short));
    }

    //allocating array of registers for magnetization and energy for inter-batch use
    hipMallocManaged(&dMagCurrent, nBeta*L*sizeof(signed short));
    hipMallocManaged(&dEnergyCurrent, nBeta*L*sizeof(signed short));

    //allocating array of cumulants for holding the sums during bootstrapping
    hipMallocManaged(&dMagCumulant, nBeta*L*nResamplings*sizeof(long long));
    hipMallocManaged(&dEnergyCumulant, nBeta*L*nResamplings*sizeof(long long));

    //allocating array of betas and lattices memory for inter-batch use
    hipMallocManaged(&dBeta, nBeta*sizeof(double));
    hipMallocManaged(&dLattices, 5400*nBeta*sizeof(bool));

    /*********************** INITIALIZATIONS ********************/
    //initialize all lattices to cold state
    for(unsigned int i=0; i<5400*nBeta; i++)
        dLattices[i]=true;

    //run random generator initialization kernel
    initRandom<<<L,nBeta>>>(dStates,1221);
    initRandom<<<L,nBeta>>>(dStatesBS, 1234);

    //create event to signal end of kernel execution
    hipEvent_t simulationDone, bootstrapDone;
    hipEventCreate(&simulationDone);
    hipEventCreate(&bootstrapDone);

    //create two streams, one for copy the other for execution of kernel
    hipStream_t Stream1, Stream2;
    hipStreamCreate(&Stream1);
    hipStreamCreate(&Stream2);
//
    /***************************** SIMULATION RUN *************************************/
    //run the kernel using one memory, while copying data to disk on the other memory
    int nBatches = dataPoints/batchSize;
    for(int i=0; i<nBatches;i++){
        if(i!=0)
        {
            //wait for kernel execution to be completed
            hipStreamWaitEvent(Stream2, simulationDone);
            bootstrap<<<L,nBeta, 0, Stream2>>>(nResamplings, batchSize, blockingSize, dMagMarkov[(i-1)%2], dEnergyMarkov[(i-1)%2], dMagCumulant, dEnergyCumulant, dStatesBS);
            hipEventRecord(bootstrapDone, Stream2);
        }
        simulation<<<L,nBeta, 0, Stream1>>>(i,batchSize, dBeta, dMagMarkov[i%2], dEnergyMarkov[i%2], dMagCurrent, dEnergyCurrent, dLattices, dStates);
        hipEventRecord(simulationDone, Stream1);
        if(i!=0)
            hipStreamWaitEvent(Stream1,bootstrapDone);
    }
    //copy the last batch produced
    hipStreamWaitEvent(Stream2, simulationDone);
    bootstrap<<<L,nBeta, 0, Stream2>>>(nResamplings, batchSize, blockingSize, dMagMarkov[(nBatches-1)%2], dEnergyMarkov[(nBatches-1)%2], dMagCumulant, dEnergyCumulant, dStatesBS);
    hipDeviceSynchronize();

    //save the results to the disk
    saveResults(dataPoints, nResamplings, nBeta, L, dMagCumulant, dEnergyCumulant);

    /********************************* CLEAN UP ****************************************/
    //destroy streams
    hipStreamDestroy(Stream1);
    hipStreamDestroy(Stream2);

    //free device memory
    hipFree(dStates);
    hipFree(dStatesBS);
    hipFree(dBeta);
    for(int i=0;i<2;i++)
    {
        hipFree(dMagMarkov[i]);
        hipFree(dEnergyMarkov[i]);
    }
    hipFree(dMagMarkov);
    hipFree(dEnergyMarkov);
    hipFree(dMagCurrent);
    hipFree(dEnergyCurrent);
    hipFree(dMagCumulant);
    hipFree(dEnergyCumulant);
    hipFree(dLattices);


    return 0;
}
